#include "hip/hip_runtime.h"
#include <iostream>

#include "GLFW\glfw3.h"
#include "glad/glad.h"

#include <hip/hip_runtime.h>
#include <>
#include <cuda_gl_interop.h>


const int N = 5;
const int TX = 32;

/// <summary>
/// Array addition parallelized in 1D.
/// </summary>
/// <param name="a">Input array 1.</param>�
/// <param name="b">Input array 2.</param>
/// <param name="out">Result.</param>
/// <param name="n">Number of elements in the arrays.</param>
/// <returns></returns>
__global__ void kernel_1D_addition(const int* a, const int* b, int* out, const int n)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i > n) return;
	out[i] = a[i] + b[i];
}
static void glfw_error_callback(int error, const char* description)
{
	fprintf(stderr, "Error: %s\n", description);
}
static void glfw_framebuffer_size_callback(GLFWwindow* window, int width, int height)
{
	glViewport(0, 0, width, height);
}
int main()
{
	// test case.
	int* a = new int[N] { 1, 1, 2, 1, 4};
	int* b = new int[N] { 1, 2, 3, 1, 1};
	int* out = new int[N] {0};
	int* d_a;
	int* d_b;
	int* d_out;
	GLFWwindow* window;
	if (!glfwInit())
		exit(EXIT_FAILURE);

	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);

	glfwSetErrorCallback(glfw_error_callback);


	window = glfwCreateWindow(1280, 720, "Cuda Application", NULL, NULL);
	if (!window)
	{
		glfwTerminate();
		exit(EXIT_FAILURE);
	}
	
	glfwMakeContextCurrent(window);
	int status = gladLoadGLLoader((GLADloadproc)glfwGetProcAddress);
	if (!status)
	{
		glfwTerminate();
		exit(EXIT_FAILURE);
	}
	glfwSwapInterval(1);

	glfwSetFramebufferSizeCallback(window, glfw_framebuffer_size_callback);

	while (!glfwWindowShouldClose(window))
	{
		glClearColor(0, 0, 256, 256);

		glfwSwapBuffers(window);
		glfwPollEvents();
	}

	glfwDestroyWindow(window);
	glfwTerminate();

	// allocating memory on the GPU
	hipMalloc(&d_a, N * sizeof(int));
	hipMalloc(&d_b, N * sizeof(int));
	hipMalloc(&d_out, N * sizeof(int));

	// copying the data from the CPU to the GPU
	hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

	// computing the grid size and block size of the kernel call 
	// NOTE: the launch parameters of the kernel has to be optimized based on the computation that has to be done
	//       There is not a proper way to calculate the best parameters, one way is simply by benchmarking.
	dim3 gridSize((N + TX - 1) / TX);
	dim3 blockSize(TX);

	// kernel call
	kernel_1D_addition << <gridSize, blockSize >> > (d_a, d_b, d_out, N);

	// waiting for all the CUDA calls to execute
	hipDeviceSynchronize();


	// copying back the data from the GPU to the CPU
	hipMemcpy(out, d_out, N * sizeof(int), hipMemcpyDeviceToHost);

	// freeing the memory
	delete[] a, b, out;
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_out);

}